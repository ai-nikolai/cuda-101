#include "hip/hip_runtime.h"
//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>


//
// kernel routine
// 

// __global__ void my_first_kernel(float *x)
// {
//   int tid = threadIdx.x + blockDim.x*blockIdx.x;
//   float y = 1/0;
//   int z = -1/0;

//   if (threadIdx.x == 1){
//     printf("y is %f, z is %d\n",y,z);
//   }

//   x[tid] = (float) y;
//   // x[tid+10000000] = (float) y;

// }


__global__ void add_kernel(float *x, float*y, float*z)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  z[tid] = x[tid]+y[tid];

}

__global__ void init_kernel(float *x, float*y, float*z)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  x[tid] = (float) tid;
  y[tid] = 2*(float) tid;

}

//
// main code
//

int main(int argc, char **argv)
{
  float *h_x, *d_x;
  float *h_y, *d_y;
  float *h_z, *d_z;
  int   nblocks, nthreads, nsize, n; 

  // set number of blocks, and threads per block

  nblocks  = 2;
  nthreads = 8;
  nsize    = nblocks*nthreads ;

  // allocate memory for array

  h_x = (float *)malloc(nsize*sizeof(float));
  checkCudaErrors(hipMalloc((void **)&d_x, nsize*sizeof(float)));

  h_y = (float *)malloc(nsize*sizeof(float));
  checkCudaErrors(hipMalloc((void **)&d_y, nsize*sizeof(float)));

  h_z = (float *)malloc(nsize*sizeof(float));
  checkCudaErrors(hipMalloc((void **)&d_z, nsize*sizeof(float)));

  // Init on host

  // for (n=0; n<nsize; n++) h_x[n] = n;
  // for (n=0; n<nsize; n++) h_y[n] = n;

  // copy init to Cuda


  // hipMemcpy(d_x,h_x,nsize*sizeof(float),hipMemcpyHostToDevice);
  // hipMemcpy(d_y,h_y,nsize*sizeof(float),hipMemcpyHostToDevice);


  // execute kernel (this call is non blocking)

  // my_first_kernel<<<nblocks,nthreads>>>(d_x);
  // my_first_custom_kernel<<<nblocks,nthreads>>>(d_x,d_y,d_z);

  init_kernel<<<nblocks,nthreads>>>(d_x,d_y,d_z);
  getLastCudaError("init_kernel execution failed\n");

  add_kernel<<<nblocks,nthreads>>>(d_x,d_y,d_z);
  getLastCudaError("add_kernel execution failed\n");

  
  // copy back results and print them out

  checkCudaErrors(hipMemcpy(h_z,d_z,nsize*sizeof(float),hipMemcpyDeviceToHost));

  for (n=0; n<nsize; n++) printf(" n,  x  =  %d  %f \n",n,h_z[n]);

  // free memory 

  hipFree(d_x);
  free(h_x);

  hipFree(d_y);
  free(h_y);

  hipFree(d_z);
  free(h_z);

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

  return 0;
}
